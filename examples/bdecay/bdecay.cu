#include "hip/hip_runtime.h"
#include "RooRealVar.h"
#include "RooDataSet.h"
#include "RooConstVar.h"
#include "RooCategory.h"
#include "RooArgSet.h"
#include "RooBDecay.h"
#include "RooFormulaVar.h"
#include "RooTruthModel.h"
#include "TApplication.h"
#include "TCanvas.h"
#include "RooPlot.h"
#include "RooDataHist.h"

//GooFit includes
#include "GooBDecayPdf.hh"
#include "FitManager.hh"

//ROOT Histogramm stuff
#include "TH1F.h"
#include <cstdio>

using namespace RooFit ;

int main()
{
    TApplication theApp("App", 0, 0);

    // C o n s t r u c t   p d f
    // -------------------------

    // Observable
    RooRealVar dt("dt","dt", 0,10) ;
    dt.setBins(40) ;

    // Parameters
    RooRealVar dm("dm","delta m(B0)",0.472) ;
    RooRealVar tau("tau","tau (B0)",1.547) ;
    RooRealVar w("w","flavour mistag rate",0.1) ;

    RooCategory tagFlav("tagFlav","Flavour of the tagged B0") ;
    tagFlav.defineType("B0",1) ;
    tagFlav.defineType("B0bar",-1) ;

    // Use delta function resolution model
    RooTruthModel tm("tm","truth model",dt) ;

    //////////////////////////////////////////////////////////////////////////////////
    // G e n e r i c   B   d e c a y  w i t h    u s e r   c o e f f i c i e n t s  //
    //////////////////////////////////////////////////////////////////////////////////

    // C o n s t r u c t   p d f
    // -------------------------

    // Model parameters
    RooRealVar DGbG("DGbG","DGamma/GammaAvg",0.5,-1,1);
    RooRealVar Adir("Adir","-[1-abs(l)**2]/[1+abs(l)**2]",0);
    RooRealVar Amix("Amix","2Im(l)/[1+abs(l)**2]",0.7);
    RooRealVar Adel("Adel","2Re(l)/[1+abs(l)**2]",0.7);

    // Derived input parameters for pdf
    RooFormulaVar DG("DG","Delta Gamma","@1/@0",RooArgList(tau,DGbG));

    // Construct coefficient functions for sin,cos,sinh modulations of decay distribution
    RooFormulaVar fsin("fsin","fsin","@0*@1*(1-2*@2)",RooArgList(Amix,tagFlav,w));
    RooFormulaVar fcos("fcos","fcos","@0*@1*(1-2*@2)",RooArgList(Adir,tagFlav,w));
    RooFormulaVar fsinh("fsinh","fsinh","@0",RooArgList(Adel));

    //GooFit wrapped variable sections, till proper
    //compatibility is established. These only include non-derived
    //variables and constants.

     //observable
    Variable goo_dt(dt.GetName(), dt.getMin(), dt.getMax());
    goo_dt.numbins = dt.getBins();

    //
    // calculate min/max for DG by hand, since there is no support
    // for RooFormulaVar

    fptype min = DGbG.getMin()/tau.getVal();
    fptype max = DGbG.getMax()/tau.getVal();
    fptype initial = DGbG.getVal()/tau.getVal();


    Variable goo_tau(tau.GetName(), tau.getVal());
    Variable goo_dm(dt.GetName(), dm.getVal());
    Variable goo_fcoshConst("cosh", 1.0);
    Variable goo_dg(DG.GetName(), initial, min, max);
    Variable goo_fsin(fsin.GetName(), fsin.getVal());
    Variable goo_fsinh(fsinh.GetName(), fsinh.getVal());
    Variable goo_fcos(fsin.GetName(), fcos.getVal());


    printf( "values: tau %.4f\n"
            "         dm %.4f\n"
            "      fcosh %.4f\n"
            "         dg %.4f, min %.4f, max %.4f\n"
            "       fsin %.4f\n"
            "      fsinh %.4f\n"
            "       fcos %.4f\n"
            , goo_tau.value
            , goo_dm.value
            , goo_fcoshConst.value
            , goo_dg.value, goo_dg.lowerlimit, goo_dg.upperlimit
            , goo_fsin.value
            , goo_fcos.value);
    fflush(stdout);


    // Construct generic B decay pdf using above user coefficients
    RooBDecay bcpg("bcpg","bcpg",dt,tau,DG,RooConst(1),fsinh,fcos,fsin,dm,tm, RooBDecay::SingleSided);



    // P l o t   -   I m ( l ) = 0 . 7 ,   R e ( l ) = 0 . 7   | l | = 1,   d G / G = 0 . 5
    // -------------------------------------------------------------------------------------

    // Generate some data
    RooDataSet* data = bcpg.generate(dt, 10000, kTRUE) ;

    bcpg.fitTo(*data);


    // GPU side RooBDecay (without any convolution = RooBDecay (**) RooTruthModel)
    GooBDecayInternal goo_bcpg("goo bdecay",
                      &goo_dt,
                      &goo_tau,
                      &goo_dg,
                      &goo_fcoshConst,
                      &goo_fsinh,
                      &goo_fcos,
                      &goo_fsin,
                      &goo_dm);

    std::vector<Variable*> vars;
    vars.push_back(&goo_dt);

    BinnedDataSet goo_data(&goo_dt);

    size_t entries = data->numEntries();

    for (size_t i = 0; i < entries; ++i) {
      const RooArgSet* args = data->get(i);
      const RooRealVar* dtArg = dynamic_cast<RooRealVar*>(args->find(dt.GetName()));

      goo_dt.value = dtArg->getVal();
      goo_data.addEvent();
    }

    //reset value to initial value
    goo_dt.value = 0;

    goo_bcpg.setData(&goo_data);

    printf("executing on gpu now\n");

    FitManager fitter(&goo_bcpg);
    fitter.fit();
    fitter.getMinuitValues();

    vector<fptype> values;
    goo_bcpg.evaluateAtPoints(&goo_dt, values);

    RooRealVar* dtClone = static_cast<RooRealVar*>(dt.Clone());
    dtClone->setVal(0.0);

    TH1D pdfHist("gpu hist", "", goo_dt.numbins, goo_dt.lowerlimit, goo_dt.upperlimit);

    double totalPdf = 0;
    for (size_t i = 0; i < values.size(); ++i) {
        totalPdf += values[i];
    }

    for (size_t i = 0; i < values.size(); ++i) {
      pdfHist.SetBinContent(i+1, values[i] / totalPdf * data->numEntries());
    }

    // Plot the generated data and both fits
    RooPlot* frame = dt.frame(Title("B decay distribution with CPV(Im(l)=0.7,Re(l)=0.7,|l|=1,dG/G=0.5) (B0/B0bar)")) ;
    RooDataHist gpuHist(pdfHist.GetName(), pdfHist.GetTitle(), RooArgSet(*dtClone), Import(pdfHist, kFALSE));

    //cpu fit in green
    data->plotOn(frame);
    bcpg.plotOn(frame, LineColor(kBlue), MarkerColor(kBlue), MarkerStyle(kBlue));
    //gpu fit in red
    gpuHist.plotOn(frame, LineColor(kRed), MarkerColor(kRed));

    TCanvas* c = new TCanvas("rf708_bphysics","rf708_bphysics",1200,800) ;
    c->cd(1) ; gPad->SetLeftMargin(0.15) ; frame->GetYaxis()->SetTitleOffset(1.6) ; frame->Draw() ;

    printf("numentries %i\n", data->numEntries());
    fflush(stdout);

    theApp.Run();
    return 0;
}
