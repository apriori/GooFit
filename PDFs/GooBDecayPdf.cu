#include "hip/hip_runtime.h"
#include "GooBDecayPdf.hh"

#include "ConvolutionPdf.hh"

EXEC_TARGET fptype device_GooBDecay(fptype* evt, fptype* p, unsigned int* indices) {
  fptype t     = evt[indices[2 + indices[0]]]; 
  fptype tau  = p[indices[1]];
  fptype dgamma = p[indices[2]];
  fptype f0 = p[indices[3]];
  fptype f1 = p[indices[4]];
  fptype f2 = p[indices[5]];
  fptype f3 = p[indices[6]];
  fptype dm = p[indices[7]];

  fptype dgt = dgamma * t /2;
  fptype dmt = dm * t;
  fptype ft = FABS(t);
  
  return exp(-ft/tau) * (f0 * cosh(dgt) 
                        +f1 * sinh(dgt)
                        +f2 * cos(dmt)
                        +f3 * sin(dmt)
                        );
}

MEM_DEVICE device_function_ptr ptr_to_BDecay = device_GooBDecay;


GooBDecayInternal::GooBDecayInternal(
    std::string n,
    Variable* t, 
    Variable* tau, 
    Variable* dgamma,
    Variable* f0,
    Variable* f1,
    Variable* f2,
    Variable* f3,
    Variable* dm
    )
  : GooPdf(t, n)
{
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(tau));
    pindices.push_back(registerParameter(dgamma));
    pindices.push_back(registerParameter(f0));
    pindices.push_back(registerParameter(f1));
    pindices.push_back(registerParameter(f2));
    pindices.push_back(registerParameter(f3));
    pindices.push_back(registerParameter(dm));
    GET_FUNCTION_ADDR(ptr_to_BDecay);
    initialise(pindices); 
}

GooBDecay::GooBDecay(std::string n,
                     Variable* t, 
                     Variable* tau, 
                     Variable* dgamma,
                     Variable* f0,
                     Variable* f1,
                     Variable* f2,
                     Variable* f3,
                     Variable* dm,
                     GooPdf* resolution
                     )
    : ConvolutionPdf(n + "_conv_" + resolution->getName(),
                     t, 
                     new GooBDecayInternal(n + "_unconv", t, tau, dgamma, f0, f1, f2, f3, dm),
                     resolution)
{
 
}



