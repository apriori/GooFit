#include "hip/hip_runtime.h"
#include "GooBDecayPdf.hh"

#include "ConvolutionPdf.hh"

EXEC_TARGET fptype device_GooBDecay(fptype* evt, fptype* p, unsigned int* indices) {
  fptype t     = evt[indices[2 + indices[0]]]; 
  fptype tag     = evt[indices[2 + indices[0] + 1]];
  fptype parS = p[indices[1]];
  fptype parC = p[indices[2]];
  fptype parOmega = p[indices[3]];
  fptype tau  = p[indices[4]];
  fptype dgamma = p[indices[5]];
  fptype f0 = p[indices[6]];
  fptype f1 = p[indices[7]];
  fptype dm = p[indices[8]];

  fptype dgt = dgamma * t /2;
  fptype dmt = dm * t;
  fptype ft = FABS(t);
  fptype coeffBase = tag * (1. - 2. * parOmega);
  fptype f2 = -coeffBase * parC;
  fptype f3 = coeffBase * parS;
  
  return exp(-ft/tau) * (f0 * cosh(dgt) 
                        +f1 * sinh(dgt)
                        +f2 * cos(dmt)
                        +f3 * sin(dmt)
                        );
}

MEM_DEVICE device_function_ptr ptr_to_BDecay = device_GooBDecay;


GooBDecayInternal::GooBDecayInternal(std::string n,
    Variable* dt,
    Variable* tag,
    Variable *parS,
    Variable *parC,
    Variable *parOmega,
    Variable* tau,
    Variable* dgamma,
    Variable* f0,
    Variable* f1,
    Variable* dm
    )
  : GooPdf(dt, n)
{
    registerObservable(tag);

    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(parS));
    pindices.push_back(registerParameter(parC));
    pindices.push_back(registerParameter(parOmega));
    pindices.push_back(registerParameter(tau));
    pindices.push_back(registerParameter(dgamma));
    pindices.push_back(registerParameter(f0));
    pindices.push_back(registerParameter(f1));
    pindices.push_back(registerParameter(dm));
    GET_FUNCTION_ADDR(ptr_to_BDecay);
    initialise(pindices); 
}

/*
GooBDecay::GooBDecay(std::string n,
                     Variable* dt,
                     Variable* tag,
                     Variable* tau, 
                     Variable* dgamma,
                     Variable* f0,
                     Variable* f1,
                     Variable* f2,
                     Variable* f3,
                     Variable* dm,
                     GooPdf* resolution
                     )
    : ConvolutionPdf(n + "_conv_" + resolution->getName(),
                     dt,
                     new GooBDecayInternal(n + "_unconv", dt, tag, tau, dgamma, f0, f1, f2, f3, dm),
                     resolution)
{
 
}
*/




