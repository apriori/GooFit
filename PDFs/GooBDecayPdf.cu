#include "hip/hip_runtime.h"
#include "GooBDecayPdf.hh"

//#include "ConvolutionPdf.hh"
#include <cstdio>

EXEC_TARGET fptype device_GooBDecay(fptype* evt, fptype* p, unsigned int* indices) {
  fptype t        = evt[indices[2 + indices[0]]];
  fptype tag      = evt[indices[2 + indices[0] + 1]];
  fptype parS     = p[indices[1]];
  fptype parC     = p[indices[2]];
  fptype parOmega = p[indices[3]];
  fptype tau      = p[indices[4]];
  fptype dgamma   = p[indices[5]];
  fptype f0       = p[indices[6]];
  fptype f1       = p[indices[7]];
  fptype dm       = p[indices[8]];

  fptype dgt = dgamma * t /2;
  fptype dmt = dm * t;
  fptype coeffBase = tag * (1. - 2. * parOmega);
  fptype f2 = -coeffBase * parC;
  fptype f3 = coeffBase * parS;
  fptype ft = FABS(t);

  fptype cosh_ = COSH(dgt);
  fptype sinh_ = SINH(dgt);
  fptype cos_ = COS(dmt);
  fptype sin_ = SIN(dmt);

  return exp(-ft/tau) * (f0 * cosh_
                        +f1 * sinh_
                        +f2 * cos_
                        +f3 * sin_
                        );
}

MEM_DEVICE device_function_ptr ptr_to_BDecay = device_GooBDecay;


GooBDecayInternal::GooBDecayInternal(std::string n,
    Variable* dt,
    Variable* tag,
    Variable *parS,
    Variable *parC,
    Variable *parOmega,
    Variable* tau,
    Variable* dgamma,
    Variable* f0,
    Variable* f1,
    Variable* dm
    )
  : GooPdf(dt, n) {
    tag->fixed = true;
    registerObservable(tag);

    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(parS));
    pindices.push_back(registerParameter(parC));
    pindices.push_back(registerParameter(parOmega));
    pindices.push_back(registerParameter(tau));
    pindices.push_back(registerParameter(dgamma));
    pindices.push_back(registerParameter(f0));
    pindices.push_back(registerParameter(f1));
    pindices.push_back(registerParameter(dm));
    GET_FUNCTION_ADDR(ptr_to_BDecay);
    initialise(pindices);
}

__host__ fptype bdecayNorm(fptype t,
                           fptype tag,
                           fptype parS,
                           fptype parC,
                           fptype parOmega,
                           fptype tau,
                           fptype f0,
                           fptype f1,
                           fptype dm) {
  fptype dmt = dm * t;
  fptype coeffBase = tag * (1. - 2. * parOmega);
  fptype f2 = -coeffBase * parC;
  fptype f3 = coeffBase * parS;

  fptype inv_coeff = (-tau)/(1 + dmt*dmt * tau * tau);
  fptype expPart = exp(-t/tau);
  fptype cosdmt = cos(dmt);
  fptype sindmt = sin(dmt);
  fptype dmttau = dmt * tau;

  fptype expIntCoeff = -tau;
  fptype sinIntCoeff = f3 * inv_coeff * (sindmt + dmttau * cosdmt);
  fptype cosIntCoeff = f2 * inv_coeff * (cosdmt - dmttau * sindmt);
  return expPart * (expIntCoeff + sinIntCoeff + cosIntCoeff);
}

fptype GooBDecayInternal::integrate(fptype lo, fptype hi) const {
  lo = std::max(lo, (fptype)0.0);
  unsigned int* indices = host_indices + parameters;
  fptype hiInt = bdecayNorm(hi,
                    1,
                    host_params[indices[1]],
                    host_params[indices[2]],
                    host_params[indices[3]],
                    host_params[indices[4]],
                    host_params[indices[6]],
                    host_params[indices[7]],
                    host_params[indices[8]]
                    );
  fptype loInt = bdecayNorm(lo,
                    1,
                    host_params[indices[1]],
                    host_params[indices[2]],
                    host_params[indices[3]],
                    host_params[indices[4]],
                    host_params[indices[6]],
                    host_params[indices[7]],
                    host_params[indices[8]]
                    );
  fptype hiInt2 = bdecayNorm(hi,
                    -1,
                     host_params[indices[1]],
                     host_params[indices[2]],
                     host_params[indices[3]],
                     host_params[indices[4]],
                     host_params[indices[6]],
                     host_params[indices[7]],
                     host_params[indices[8]]
                    );
  fptype loInt2 = bdecayNorm(lo,
                    -1,
                   host_params[indices[1]],
                   host_params[indices[2]],
                   host_params[indices[3]],
                   host_params[indices[4]],
                   host_params[indices[6]],
                   host_params[indices[7]],
                   host_params[indices[8]]
                    );
  //return hiInt2 + hiInt - loInt - loInt2;
  fptype ttau = host_params[indices[4]];
  return 2*-ttau * (exp(-hi/ttau) - exp(-lo/ttau));
  //return hiInt - loInt;
}

/*
GooBDecay::GooBDecay(std::string n,
                     Variable* dt,
                     Variable* tag,
                     Variable* tau, 
                     Variable* dgamma,
                     Variable* f0,
                     Variable* f1,
                     Variable* f2,
                     Variable* f3,
                     Variable* dm,
                     GooPdf* resolution
                     )
    : ConvolutionPdf(n + "_conv_" + resolution->getName(),
                     dt,
                     new GooBDecayInternal(n + "_unconv", dt, tag, tau, dgamma, f0, f1, f2, f3, dm),
                     resolution)
{
 
}
*/




