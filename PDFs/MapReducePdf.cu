#include "MapReducePdf.hh"

#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
#include <thrust/system/cuda/detail/bulk.h>
using namespace thrust::system::cuda::detail;
struct ComponentsParallelEval {
  ComponentsParallelEval(size_t numEvents,
                         size_t numObs)
   : numEvents(numEvents)
   , numObs(numObs) {

  }

  EXEC_TARGET
  void operator()(bulk_::agent<> &self,
                  thrust::device_ptr<fptype> y,
                  size_t component,
                  size_t function,
                  size_t params,
                  thrust::constant_iterator<fptype*> events
                  ) {
    size_t offsetResult = self.index() + component * numEvents;
    fptype* eventAddress = *events + self.index() * numObs;
    y[offsetResult] = callFunction(eventAddress, function, params);
  }
  size_t numEvents;
  size_t numObs;
};
#endif

MapReducePdf::MapReducePdf(std::string n,
                           std::vector<PdfBase*> comps,
                           std::vector<size_t> extraIndices,
                           std::vector<fptype> extraDoubles,
                           std::vector<Variable*> extraParams)
 : GooPdf(0, n) {
  assert(comps.size() != 0);

  indices.push_back(comps.size());
  indices.push_back(extraIndices.size());
  indices.push_back(extraDoubles.size());
  indices.push_back(extraParams.size());
  // reserved for later initialisation by preEvaulateComponents
  // this index is supposed to be actually a device side address
  // to the device vector containing precalculated values of
  // the components
  componentValuesAddressParamIndex = indices.size();
  indices.push_back(0);
  // reserved for later initialisation by preEvaulateComponents
  // meant to contain the address of the start of dev_event_array
  eventArrayAddressParamIndex = indices.size();
  indices.push_back(0);
  // reserved for later initilation by preEvaulateComponents
  // meant to contain the number of events to be calculated.
  // this is needed for in-kernel pointer arithmetic
  numEventsParamIndex = indices.size();
  indices.push_back(0);

  for (size_t i = 0; i < comps.size(); ++i) {
    hipStream_t stream;
    hipStreamCreate(&stream);

    streams.push_back(stream);
    components.push_back(comps[i]);
    indices.push_back(comps[i]->getFunctionIndex());
    indices.push_back(comps[i]->getParameterIndex());
  }

  for (size_t i = 0; i < extraIndices.size(); ++i) {
    indices.push_back(extraIndices[i]);
  }

  for (size_t i = 0; i < extraDoubles.size(); ++i) {
    indices.push_back(extraDoubles[i]);
  }

  for (size_t i = 0; i < extraParams.size(); ++i) {
    indices.push_back(registerParameter(extraParams[i]));
  }
}

MapReducePdf::~MapReducePdf() {
#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
  for (size_t i = 0; i < components.size(); ++i) {
    hipStreamDestroy(streams[i]);
  }
#endif
}

void MapReducePdf::delayedInitialize() {
  initialise(indices);
}

void MapReducePdf::onDataChanged(size_t numEvents) {
  this->numEvents = numEvents;
  std::cout << "ondata changed " << getName()  << std::endl;
#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
  componentValues = thrust::device_vector<fptype>(components.size() * numEvents);
  size_t* indices = host_indices + parameters + 1;
  indices[numEventsParamIndex] = numEvents;
  indices[eventArrayAddressParamIndex] = (size_t)dev_event_array;
  indices[componentValuesAddressParamIndex] = (size_t)thrust::raw_pointer_cast(componentValues.data());
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams*sizeof(unsigned long), 0, hipMemcpyHostToDevice);
#endif
}

void MapReducePdf::preEvaluateComponents(std::vector<bulk_::future<void> >& futures) const {
  //std::cout << "pre eval " << getName() << std::endl;
#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
  if (numEvents == 0) {
    return;
  }

  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array);

  ComponentsParallelEval eval(numEvents, observables.size());
  for (size_t i = 0; i < components.size(); ++i) {
    PdfBase* pdf = components[i];

    futures.push_back(
      bulk_::async(bulk_::par(streams[i], numEvents),
                   eval,
                   bulk_::root.this_exec,
                   componentValues.data(),
                   i,
                   pdf->getFunctionIndex(),
                   pdf->getParameterIndex(),
                   arrayAddress));
  }
#endif
}
