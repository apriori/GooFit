#include "hip/hip_runtime.h"
#include "AddPdf.hh"

#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/transform_iterator.h>

#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
EXEC_TARGET fptype device_AddPdfs (fptype* evt, fptype* p, unsigned long* indices) {
  long numParameters = indices[0];
  fptype ret = 0;
  fptype totalWeight = 0;
  for (long i = 1; i < numParameters-3; i += 3) {
    totalWeight += p[indices[i+2]];
    fptype curr = callFunction(evt, indices[i], indices[i+1]);
    fptype weight = p[indices[i+2]];
    ret += weight * curr * normalisationFactors[indices[i+1]];

    //if ((gpuDebug & 1) && (0 == THREADIDX) && (0 == BLOCKIDX))
    //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    //printf("Add comp %i: %f * %f * %f = %f (%f)\n", i, weight, curr, normalisationFactors[indices[i+1]], weight*curr*normalisationFactors[indices[i+1]], ret);

  }
  // numParameters does not count itself. So the array structure for two functions is
  // nP | F P w | F P
  // in which nP = 5. Therefore the parameter index for the last function pointer is nP, and the function index is nP-1.
  //fptype last = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[numParameters-1]])))(evt, p, paramIndices + indices[numParameters]);
  fptype last = callFunction(evt, indices[numParameters - 1], indices[numParameters]);
  ret += (1 - totalWeight) * last * normalisationFactors[indices[numParameters]];

  //if ((THREADIDX < 50) && (isnan(ret))) printf("NaN final component %f %f\n", last, totalWeight);

  //if ((gpuDebug & 1) && (0 == THREADIDX) && (0 == BLOCKIDX))
  //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
  //printf("Add final: %f * %f * %f = %f (%f)\n", (1 - totalWeight), last, normalisationFactors[indices[numParameters]], (1 - totalWeight) *last* normalisationFactors[indices[numParameters]], ret);

  return ret;
}

EXEC_TARGET fptype device_AddPdfsExt (fptype* evt, fptype* p, unsigned long* indices) {
  // numParameters does not count itself. So the array structure for two functions is
  // nP | F P w | F P w
  // in which nP = 6.

  long numParameters = indices[0];
  fptype ret = 0;
  fptype totalWeight = 0;
  for (long i = 1; i < numParameters; i += 3) {
    //fptype curr = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[i]])))(evt, p, paramIndices + indices[i+1]);
    fptype curr = callFunction(evt, indices[i], indices[i+1]);
    fptype weight = p[indices[i+2]];
    ret += weight * curr * normalisationFactors[indices[i+1]];

    totalWeight += weight;
    //if ((gpuDebug & 1) && (THREADIDX == 0) && (0 == BLOCKIDX))
    //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    //printf("AddExt: %i %E %f %f %f %f %f %f\n", i, curr, weight, ret, totalWeight, normalisationFactors[indices[i+1]], evt[0], evt[8]);
  }
  ret /= totalWeight;
  //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
  //if ((gpuDebug & 1) && (THREADIDX == 0) && (0 == BLOCKIDX))
  //printf("AddExt result: %f\n", ret);

  return ret;
}


#else
#include <thrust/system/cuda/detail/bulk.h>
using namespace thrust::system::cuda::detail;
EXEC_TARGET fptype device_AddPdfs (fptype* evt, fptype* p, unsigned long* indices) {
  /*
  int components = indices[0];
  fptype ret = 0;
  fptype totalWeight = 0;
  for (int i = 1; i < numParameters-3; i += 3) {
    totalWeight += p[indices[i+2]];
    //fptype curr = callFunction(evt, indices[i], indices[i+1]);
    fptype weight = p[indices[i+2]];
    ret += weight * curr * normalisationFactors[indices[i+1]];

    //if ((gpuDebug & 1) && (0 == THREADIDX) && (0 == BLOCKIDX))
    //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    //printf("Add comp %i: %f * %f * %f = %f (%f)\n", i, weight, curr, normalisationFactors[indices[i+1]], weight*curr*normalisationFactors[indices[i+1]], ret);

  }
  // numParameters does not count itself. So the array structure for two functions is
  // nP | F P w | F P
  // in which nP = 5. Therefore the parameter index for the last function pointer is nP, and the function index is nP-1.
  //fptype last = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[numParameters-1]])))(evt, p, paramIndices + indices[numParameters]);
  //fptype last = callFunction(evt, indices[numParameters - 1], indices[numParameters]);
  ret += (1 - totalWeight) * last * normalisationFactors[indices[numParameters]];

  //if ((THREADIDX < 50) && (isnan(ret))) printf("NaN final component %f %f\n", last, totalWeight);

  //if ((gpuDebug & 1) && (0 == THREADIDX) && (0 == BLOCKIDX))
  //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
  //printf("Add final: %f * %f * %f = %f (%f)\n", (1 - totalWeight), last, normalisationFactors[indices[numParameters]], (1 - totalWeight) *last* normalisationFactors[indices[numParameters]], ret);
  */

  return 0;
  //return ret;
}

EXEC_TARGET fptype device_AddPdfsExt (fptype* evt, fptype* p, size_t* indices) {
  size_t components = indices[1];
  size_t valueStartAddress = indices[2];
  size_t eventStartAddress = indices[3];
  size_t numEvents = indices[4];
  size_t numObs = indices[indices[0]+1];
  size_t eventIndex = (size_t)(evt - (fptype*)eventStartAddress)/numObs;
  fptype* valueStart = reinterpret_cast<fptype*>(valueStartAddress);

  printf("addr valuestart is %lx\n", valueStart);
  //printf("numobs is %lu\n", numObs);
  printf("evt index is %lu\n", (size_t)eventIndex);
  //printf("components in gpu %lu\n", components);
  //printf("event addr in gpu %lx\n", eventStartAddress);
  //printf("value addr in gpu %lx\n", valueStartAddress);
  //printf("numEvents in gpu %lu\n", numEvents);

  fptype ret = 0;
  fptype totalWeight = 0;
  for (size_t i = 0; i < components; i ++) {
    size_t inComponentValueIndex = i * numEvents + eventIndex;
    //printf("in comp idx %lu\n", inComponentValueIndex);
    fptype curr = valueStart[inComponentValueIndex];
    //printf("curr is %f\n", curr);

    fptype weight = p[indices[i+4]];
    ret += weight * curr * normalisationFactors[indices[i+1]];

    totalWeight += weight;
    //if ((gpuDebug & 1) && (THREADIDX == 0) && (0 == BLOCKIDX))
    //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    //printf("AddExt: %i %E %f %f %f %f %f %f\n", i, curr, weight, ret, totalWeight, normalisationFactors[indices[i+1]], evt[0], evt[8]);
  }
  ret /= totalWeight;
  //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
  //if ((gpuDebug & 1) && (THREADIDX == 0) && (0 == BLOCKIDX))
  //printf("AddExt result: %f\n", ret);

  return ret;
}
#endif

MEM_DEVICE device_function_ptr ptr_to_AddPdfs = device_AddPdfs; 
MEM_DEVICE device_function_ptr ptr_to_AddPdfsExt = device_AddPdfsExt; 

AddPdf::AddPdf (std::string n, std::vector<Variable*> weights, std::vector<PdfBase*> comps) 
  : GooPdf(0, n)
  , componentValues(0)
  , eventArrayAddressParamIndex(-1)
  , componentValuesAddressParamIndex(-1)
  , numEventsParamIndex(-1)
  , weights(weights)
  , extended(true) {

  assert((weights.size() == comps.size()) || (weights.size() + 1 == comps.size())); 

  // Indices stores (components count)(component values start)(weight index_1, ... index_n) tuple
  // Last component has no weight index unless function is extended. 
  for (std::vector<PdfBase*>::iterator p = comps.begin(); p != comps.end(); ++p) {
    components.push_back(*p); 
    assert(components.back()); 
  }

  getObservables(observables); 

  std::vector<unsigned long> pindices;

  std::cout << "init components " << components.size() << std::endl;
#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  for (size_t w = 0; w < weights.size(); ++w) {
    assert(components[w]);
    pindices.push_back(components[w]->getFunctionIndex());
    pindices.push_back(components[w]->getParameterIndex());
    pindices.push_back(registerParameter(weights[w]));
  }
  assert(components.back());
  if (weights.size() < components.size()) {
    pindices.push_back(components.back()->getFunctionIndex());
    pindices.push_back(components.back()->getParameterIndex());
    extended = false;
  }
#else
  pindices.push_back(components.size());
  // reserved for later initialisation by sumOfNLL
  // this index is supposed to be actually a device side address
  // to the device vector containing precalculated values of
  // the components
  componentValuesAddressParamIndex = pindices.size();
  pindices.push_back(0);
  // reserved for later initialisation by sumOfNLL
  // meant to contain the address of the start of dev_event_array
  eventArrayAddressParamIndex = pindices.size();
  pindices.push_back(0);
  // reserved for later initilation by sumOfNLL
  // meant to contain the number of events to be calculated
  numEventsParamIndex = pindices.size();
  pindices.push_back(0);

  for (unsigned int w = 0; w < weights.size(); ++w) {
    assert(components[w]);
    pindices.push_back(registerParameter(weights[w])); 
  }
  assert(components.back()); 
  if (weights.size() < components.size()) {
    extended = false; 
  }
#endif

  if (extended) GET_FUNCTION_ADDR(ptr_to_AddPdfsExt);
  else GET_FUNCTION_ADDR(ptr_to_AddPdfs);

  initialise(pindices); 
} 


AddPdf::AddPdf (std::string n, Variable* frac1, PdfBase* func1, PdfBase* func2) 
  : GooPdf(0, n)
  , componentValues(0)
  , eventArrayAddressParamIndex(-1)
  , componentValuesAddressParamIndex(-1)
  , numEventsParamIndex(-1)
  , extended(false)
{
  // Special-case constructor for common case of adding two functions.
  components.push_back(func1);
  components.push_back(func2);
  getObservables(observables); 

  std::vector<unsigned long> pindices;
  pindices.push_back(func1->getFunctionIndex());
  pindices.push_back(func1->getParameterIndex());
  pindices.push_back(registerParameter(frac1)); 

  pindices.push_back(func2->getFunctionIndex());
  pindices.push_back(func2->getParameterIndex());
    
  GET_FUNCTION_ADDR(ptr_to_AddPdfs);

  initialise(pindices);
}

__host__ fptype AddPdf::normalise () const {
  //if (cpuDebug & 1) std::cout << "Normalising AddPdf " << getName() << std::endl;
  fptype ret = 0;
  fptype totalWeight = 0;

#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  for (unsigned int i = 0; i < components.size()-1; ++i) {
    fptype weight = host_params[host_indices[parameters + 3*(i+1)]];
    totalWeight += weight;
    fptype curr = components[i]->normalise();
    ret += curr*weight;
  }
  fptype last = components.back()->normalise();
  if (extended) {
    fptype lastWeight = host_params[host_indices[parameters + 3*components.size()]];
    totalWeight += lastWeight;
    ret += last * lastWeight;
    ret /= totalWeight;
  }
  else {
    ret += (1 - totalWeight) * last;
  }
#else
  for (ptrdiff_t i = 0; i < components.size()-1; ++i) {
    fptype weight = host_params[host_indices[parameters + numEventsParamIndex + 1 + i]];
    totalWeight += weight;
    fptype curr = components[i]->normalise();
    ret += curr*weight;
  }
  fptype last = components.back()->normalise();
  if (extended) {
    fptype lastWeight = host_params[host_indices[parameters + numEventsParamIndex + 1 + components.size()]];
    totalWeight += lastWeight;
    ret += last * lastWeight;
    ret /= totalWeight;
  }
  else {
    ret += (1 - totalWeight) * last;
  }
#endif
  host_normalisation[parameters] = 1.0;
  if (getSpecialMask() & PdfBase::ForceCommonNorm) {
    // Want to normalise this as
    // (f1 A + (1-f1) B) / int (f1 A + (1-f1) B)
    // instead of default
    // (f1 A / int A) + ((1-f1) B / int B).

    for (unsigned int i = 0; i < components.size(); ++i) {
      host_normalisation[components[i]->getParameterIndex()] = (1.0 / ret);
    }
  }

  //if (cpuDebug & 1) std::cout << getName() << " integral returning " << ret << std::endl; 
  return ret; 
}

#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
struct AddPdfEval {
  AddPdfEval(size_t xBound)
   : xBound(xBound) {

  }

  EXEC_TARGET
  void operator()(bulk_::agent<> &self,
                  thrust::device_ptr<fptype> y,
                  thrust::device_ptr<thrust::tuple<int, int> > fIdx,
                  thrust::constant_iterator<fptype*> events
                  ) {
    size_t component = self.index() / xBound;
    size_t offset = self.index() % xBound;
    thrust::tuple<int, int> functionTuple = fIdx.get()[component];
    int function = thrust::get<0>(functionTuple);
    int params = thrust::get<1>(functionTuple);
    fptype* eventAddress = *events + offset;
    y[self.index()] = callFunction(eventAddress, function, params);
    //y[self.index()] = 0;
  }

  size_t xBound;
};
#endif

__host__ double AddPdf::sumOfNll (int numVars) const {
#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
  preEvaluateComponents();
#endif

  static thrust::plus<double> cudaPlus;
  thrust::constant_iterator<int> eventSize(numVars);
  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array);
  double dummy = 0;
  thrust::counting_iterator<int> eventIndex(0);
  double ret = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
                    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                    *logger, dummy, cudaPlus);

  if (extended) {
    fptype expEvents = 0;
    //std::cout << "Weights:";
    for (unsigned int i = 0; i < components.size(); ++i) {
      expEvents += host_params[host_indices[parameters + 3*(i+1)]];
      //std::cout << " " << host_params[host_indices[parameters + 3*(i+1)]];
    }
    // Log-likelihood of numEvents with expectation of exp is (-exp + numEvents*ln(exp) - ln(numEvents!)).
    // The last is constant, so we drop it; and then multiply by minus one to get the negative log-likelihood.
    ret += (expEvents - numEvents*log(expEvents));
    //std::cout << " " << expEvents << " " << numEvents << " " << (expEvents - numEvents*log(expEvents)) << std::endl;
  }

  std::cout << "returning " << ret << std::endl;
  return ret;
}

#if THRUST_DEVICE_SYSTEM!=THRUST_DEVICE_BACKEND_OMP
void AddPdf::preEvaluateComponents() const {
  if (numEntries == 0) {
    return;
  }

  std::cout << "vector size is " << components.size() * numEntries << std::endl;

  if (componentValues) {
    delete componentValues;
  }

  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array);
  componentValues = new thrust::device_vector<fptype>(components.size() * numEntries);
  thrust::device_vector<thrust::tuple<int, int> > functionAndParamIndices(components.size());
  thrust::host_vector<thrust::tuple<int, int> > functionAndParamIndices_host(components.size());

  for (unsigned int i = 0; i < components.size(); ++i) {
    PdfBase* pdf = components[i];
    functionAndParamIndices_host[i] = thrust::make_tuple(pdf->getFunctionIndex(),
                                                         pdf->getParameterIndex());
  }
  functionAndParamIndices = functionAndParamIndices_host;

  AddPdfEval eval(numEntries);
  bulk_::async(bulk_::par(components.size() * numEntries),
              eval,
              bulk_::root.this_exec,
              componentValues->data(),
              functionAndParamIndices.data(),
              arrayAddress).wait();

  thrust::host_vector<fptype> hval = *componentValues;

  for (int i = 0; i < 100; ++i) {
    std::cout << "val " << i << " : " << hval[i] << std::endl;
  }

  size_t* indices = host_indices + parameters + 1;
  indices[numEventsParamIndex] = numEntries;
  indices[eventArrayAddressParamIndex] = (size_t)&dev_event_array[0];
  indices[componentValuesAddressParamIndex] = (size_t)thrust::raw_pointer_cast(componentValues->data());

  std::cout << "compidx " << 0 << std::endl;
  std::cout << "evidx " << eventArrayAddressParamIndex << std::endl;
  std::cout << "validx " << componentValuesAddressParamIndex << std::endl;
  std::cout << "numidx " << numEventsParamIndex << std::endl;

  std::cout << std::hex << "host components " << indices[0] << std::endl;
  std::cout << std::hex << "host event addr " << (size_t)&dev_event_array[0] << std::endl;
  std::cout << std::hex << "host value addr " << (size_t)thrust::raw_pointer_cast(componentValues->data()) << std::endl;
  std::cout << std::dec << "host num events " << indices[numEventsParamIndex] << std::endl;

  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams*sizeof(unsigned long), 0, hipMemcpyHostToDevice);
}
#endif

