#include "hip/hip_runtime.h"
#include "MapReduceAddPdf.hh"

EXEC_TARGET fptype device_MapReduceAddPdfs (fptype* evt, fptype* p, unsigned long* indices) {
  size_t valueStartAddress = indices[1];
  size_t eventStartAddress = indices[2];
  size_t numEvents = indices[3];
  size_t numObs = indices[indices[0]+1];
  size_t eventIndex = (size_t)(evt - (fptype*)eventStartAddress)/numObs;
  size_t pIndexStart = 4;
  fptype* valueStart = reinterpret_cast<fptype*>(valueStartAddress);
  fptype ret = 0;
  fptype totalWeight = 0.0;
  size_t inComponentValueIndex = eventIndex;
  fptype curr = valueStart[inComponentValueIndex];
  fptype weight = p[indices[pIndexStart + 1]];

  ret += weight * curr * normalisationFactors[indices[pIndexStart]];
  inComponentValueIndex += numEvents;
  curr = valueStart[inComponentValueIndex];
  ret += (1 - totalWeight) * curr * normalisationFactors[indices[pIndexStart + 2]];
  return ret;
}

EXEC_TARGET fptype device_MapReduceAddPdfsExt (fptype* evt, fptype* p, size_t* indices) {
  size_t components = indices[1];
  size_t valueStartAddress = indices[5];
  size_t eventStartAddress = indices[6];
  size_t numEvents = indices[7];
  size_t numObs = indices[indices[0]+1];
  size_t eventIndex = (size_t)(evt - (fptype*)eventStartAddress)/numObs;
  size_t pIndexStart = 8;
  size_t weightStart = pIndexStart + components * 2;
  fptype* valueStart = reinterpret_cast<fptype*>(valueStartAddress);

  fptype ret = 0;
  fptype totalWeight = 0;
  for (size_t i = 0; i < components; i++) {
    size_t inComponentValueIndex = i * numEvents + eventIndex;
    fptype curr = valueStart[inComponentValueIndex];
    fptype weight = p[indices[weightStart + i]];

    //printf("curr %f weight %f norm %f\n", curr, weight, normalisationFactors[indices[pIndexStart + 2 * i + 1]]);

    ret += weight * curr * normalisationFactors[indices[pIndexStart + 2 * i + 1]];
    totalWeight += weight;

    //if ((gpuDebug & 1) && (THREADIDX == 0) && (0 == BLOCKIDX))
    //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    //printf("AddExt: %i %E %f %f %f %f %f %f\n", i, curr, weight, ret, totalWeight, normalisationFactors[indices[i+1]], evt[0], evt[8]);
  }

  ret /= totalWeight;
  //if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
  //if ((gpuDebug & 1) && (THREADIDX == 0) && (0 == BLOCKIDX))
  //printf("AddExt result: %f\n", ret);

  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_MapReduceAddPdfs = device_MapReduceAddPdfs;
MEM_DEVICE device_function_ptr ptr_to_MapReduceAddPdfsExt = device_MapReduceAddPdfsExt;

MapReduceAddPdf::MapReduceAddPdf (std::string n, std::vector<Variable*> weights, std::vector<PdfBase*> comps)
  : MapReducePdf(n,
                 comps,
                 std::vector<size_t>(),
                 std::vector<fptype>(),
                 weights) {

  assert((weights.size() == comps.size()) || (weights.size() + 1 == comps.size()));
  getObservables(observables);
  GET_FUNCTION_ADDR(ptr_to_MapReduceAddPdfsExt);
  delayedInitialize();
}

__host__ fptype MapReduceAddPdf::normalise() const {
  //if (cpuDebug & 1) std::cout << "Normalising AddPdf " << getName() << std::endl;
  fptype ret = 0;
  fptype totalWeight = 0;

#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  for (unsigned int i = 0; i < components.size()-1; ++i) {
    fptype weight = host_params[host_indices[parameters + 3*(i+1)]];
    totalWeight += weight;
    fptype curr = components[i]->normalise();
    ret += curr*weight;
  }
  fptype last = components.back()->normalise();
  if (extended) {
    fptype lastWeight = host_params[host_indices[parameters + 3*components.size()]];
    totalWeight += lastWeight;
    ret += last * lastWeight;
    ret /= totalWeight;
  }
  else {
    ret += (1 - totalWeight) * last;
  }
#else
  size_t weightStartIndex = parameters + numEventsParamIndex + 1 + 2 * components.size();
  for (ptrdiff_t i = 0; i < components.size()-1; ++i) {
    fptype weight = host_params[host_indices[weightStartIndex + i]];
    totalWeight += weight;
    fptype curr = components[i]->normalise();
    ret += curr*weight;
  }
  fptype last = components.back()->normalise();
  if (true) {
    fptype lastWeight = host_params[host_indices[weightStartIndex + components.size() -1]];
    totalWeight += lastWeight;
    ret += last * lastWeight;
    ret /= totalWeight;
  }
  else {
    ret += (1 - totalWeight) * last;
  }
#endif
  host_normalisation[parameters] = 1.0;
  if (getSpecialMask() & PdfBase::ForceCommonNorm) {
    // Want to normalise this as
    // (f1 A + (1-f1) B) / int (f1 A + (1-f1) B)
    // instead of default
    // (f1 A / int A) + ((1-f1) B / int B).

    for (unsigned int i = 0; i < components.size(); ++i) {
      host_normalisation[components[i]->getParameterIndex()] = (1.0 / ret);
    }
  }

  if (cpuDebug & 1) std::cout << getName() << " integral returning " << ret << std::endl;
  return ret;
}

__host__ double MapReduceAddPdf::sumOfNll(int numVars) const {
  recursivePreEvaluateComponents();

  static thrust::plus<double> cudaPlus;
  thrust::constant_iterator<int> eventSize(numVars);
  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array);
  double dummy = 0;
  thrust::counting_iterator<int> eventIndex(0);
  double ret = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
                    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                    *logger, dummy, cudaPlus);

  if (true) {
    fptype expEvents = 0;
    for (size_t i = 0; i < components.size(); ++i) {
#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
      expEvents += host_params[host_indices[parameters + 3*(i+1)]];
#else
      size_t weightStartIndex = parameters + numEventsParamIndex + 1 + 2 * components.size();
      expEvents += host_params[host_indices[weightStartIndex + i]];
#endif
    }
    // Log-likelihood of numEvents with expectation of exp is (-exp + numEvents*ln(exp) - ln(numEvents!)).
    // The last is constant, so we drop it; and then multiply by minus one to get the negative log-likelihood.
    ret += (expEvents - numEvents*log(expEvents));
    //std::cout << " " << expEvents << " " << numEvents << " " << (expEvents - numEvents*log(expEvents)) << std::endl;
  }

  //std::cout << "returning " << ret << std::endl;
  exit(1);
  return ret;
}
