#include "hip/hip_runtime.h"
#include "MapReduceProdPdf.hh"

EXEC_TARGET fptype device_MapReduceProdPdf (fptype* evt, fptype* p, unsigned long* indices) {
  size_t components = indices[1];
  size_t valueStartAddress = indices[5];
  size_t eventStartAddress = indices[6];
  size_t numEvents = indices[7];
  size_t numObs = indices[indices[0]+1];
  size_t eventIndex = (size_t)(evt - (fptype*)eventStartAddress)/numObs;
  fptype* valueStart = reinterpret_cast<fptype*>(valueStartAddress);
  size_t pIndexStart = 8;

  fptype ret = 1;
  for (size_t i = 0; i < components; i++) {
    size_t inComponentValueIndex = i * numEvents + eventIndex;
    fptype curr = valueStart[inComponentValueIndex];

    printf("curr %f norm %f\n", curr, normalisationFactors[indices[pIndexStart + 2 * i + 1]]);

    ret *= normalisationFactors[indices[pIndexStart + 2 * i + 1]];
    ret *= curr;
  }
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_MapReduceProdPdf = device_MapReduceProdPdf;

MapReduceProdPdf::MapReduceProdPdf(std::string n,
                                   std::vector<PdfBase*> comps)
  : MapReducePdf(n, comps) {

  getObservables(observables); // Gathers from components

  PdfBase::obsCont observableCheck; // Use to check for overlap in observables

  for (std::vector<PdfBase*>::iterator p = comps.begin(); p != comps.end(); ++p) {
    if (varOverlaps) continue; // Only need to establish this once.
    PdfBase::obsCont currObses;
    (*p)->getObservables(currObses);
    for (PdfBase::obsIter o = currObses.begin(); o != currObses.end(); ++o) {
      if (find(observableCheck.begin(), observableCheck.end(), (*o)) == observableCheck.end()) continue;
      varOverlaps = true;
      break;
    }
    (*p)->getObservables(observableCheck);
  }

  if (varOverlaps) { // Check for components forcing separate normalisation
    for (std::vector<PdfBase*>::iterator p = comps.begin(); p != comps.end(); ++p) {
      if ((*p)->getSpecialMask() & PdfBase::ForceSeparateNorm) varOverlaps = false;
    }
  }
  GET_FUNCTION_ADDR(ptr_to_MapReduceProdPdf);
  delayedInitialize();
}

fptype MapReduceProdPdf::normalise() const {
  if (varOverlaps) {
    // Two or more components share an observable and cannot be separately
    // normalised, since \int A*B dx does not equal int A dx * int B dx.
    recursiveSetNormalisation(fptype(1.0));
    MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice);

    // Normalise numerically.
    //std::cout << "Numerical normalisation of " << getName() << " due to varOverlaps.\n";
    fptype ret = GooPdf::normalise();
    //if (cpuDebug & 1)
    //std::cout << "ProdPdf " << getName() << " has normalisation " << ret << " " << host_callnumber << std::endl;
    return ret;
  }

  // Normalise components individually
  for (std::vector<PdfBase*>::const_iterator c = components.begin(); c != components.end(); ++c) {
    (*c)->normalise();
  }
  host_normalisation[parameters] = 1;
  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice);

  return 1.0;
}
