#include "hip/hip_runtime.h"
#include "GooDecayPdf.hh"

EXEC_TARGET fptype device_GooDecay (fptype* evt, fptype* p, unsigned long* indices) {
  fptype t = evt[indices[2 + indices[0]]]; 
  fptype tau = p[indices[1]];
  fptype ret = EXP(-t/tau);
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_GooDecay = device_GooDecay; 

__host__ GooDecayPdf::GooDecayPdf (std::string n, Variable* _t, Variable* tau)
  : GooPdf(_t, n) 
{
  std::vector<unsigned long> pindices;
  pindices.push_back(registerParameter(tau));
  GET_FUNCTION_ADDR(ptr_to_GooDecay);
  initialise(pindices); 
}

__host__ fptype GooDecayPdf::integrate (fptype lo, fptype hi) const {
  lo = std::max(lo, (fptype)0.0);
  unsigned long* indices = host_indices + parameters;
  fptype tau = host_params[indices[1]];
  return -tau * (EXP(-hi/tau) - EXP(-lo/tau));
}

